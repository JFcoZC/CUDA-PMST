#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdlib.h>

//VARIABLES GLOBALES
#define NUMVERTICES 10
#define MAXTRHEADSXBLOCK 32

//ID gpudevice that is used
int gpudev = 0;

//Graph representation with
int *EG;    //Double array of edges |NUMVERTICES|
int *VG;    //Double array of vertices

int C;      //Current vertex INDEX
int NUMBEREDGES;

//MST edge list: Shows the path that is followed.
int *R1source;
int *R2destination;
int *R3weigth;

//Temporal arrays used for reduction results
int *T1weights;
int *T2indexes;

//------- FUNCIONES --------
void printDoubleArray(int *VX)
{
    int lengthArray = NUMVERTICES;

    for(int i = 0; i <lengthArray; i++)
    {
        printf("%i ", VX[i]);

    }//End for 2

    printf("\n");

    for(int i = lengthArray; i < (lengthArray*2); i++)
    {
         printf("%i ",VX[i]);

    }//End for 2

    printf("\n");

}//Fin funcion printDoubleArray
//------------------------
void printArrayRange(int *VX,int start,int end)
{

    for(int i = start; i <= end; i++)
    {
        printf("%2i ", VX[i]);

    }//End for 2

    printf("\n");

}//Fin funcion printArrayRange
//------------------------
/*
*Function that creates the vaules of the graph
*in the strucutre.
*/
void setGraph()
{
    //Asign and initialize memory for the double array of integers
    //Because is a doble array it is multiplied by 2 only
    VG = (int *) calloc(NUMVERTICES*2, NUMVERTICES*2*sizeof(int) );
    
    int numberEdges = 0;
    int randValue = 0;

    //Inicializacion valores VG
    for (int i = 0; i < NUMVERTICES; i++)
    {
        //Set the index of the Vertex
        VG[i] = i;
        //Set in random way the # of vertices to 
        //wich this vertex is connected

        //#Of vertices can not be 0,becuase all the veritces
        //have to be connected so ensure that at least all
        // the nodes are connected to at leas 2 other vertices
        randValue = rand() % (NUMVERTICES-2) +2;
        VG[i+NUMVERTICES] = randValue;
        //Keep track of the number edges
        numberEdges = numberEdges + randValue;

    }//Fin for 1

    //!!!SAVE IN GLOBAL VARIABLE!!!
    NUMBEREDGES = numberEdges

    //----------------
    printf("-- Source Vertex --\n");
    printDoubleArray(VG);
    printf("------\n");
    printf("TOTAL EDGES: %i\n",numberEdges);
    //----------------

    //Asign and initialize memory for the double array of integers
    //Because is a doble array it is multiplied by 2 only
    EG = (int *) calloc(numberEdges*2,numberEdges*2*sizeof(int));

    //Initialize EDGE Double array values
    int indxEdges = 0;

    for(int i = 0; i < NUMVERTICES; i++)
    {
        //Num of vertices to wich vertex i has a path
        int numVerticesConn = VG[i+NUMVERTICES];

        //1)Set the destinatio id of the vertex, which can not
        //be repeated and can not be the same as the source
        //vertex i
        int indxDestination = 0;

        //2)Set randomly the value of the weight of edge 1)
        //values of weight from 1 - 100 
        for(int j = 0; j < numVerticesConn; j++)
        {
            //1)
            //Ojo: indxDestination = j a menos que se encuentr
            //que source = destino; en ese caso y por el resto
            //del for j, indxDestination ira uno arriba que j
            if( i == j )
            {
                indxDestination++;
            }//End if

            EG[indxEdges] = indxDestination;

            //2)
            EG[indxEdges+numberEdges] = rand() % (100) +1;

            indxEdges++;
            indxDestination++; 

        }//End for 3

    }//Fin for 2

    //----------------
    printf("-- Destination vertex --\n");
    printArrayRange(EG,0,numberEdges-1);
    printf("-- Weigth of Edge --\n");
    printArrayRange(EG,numberEdges,(numberEdges*2)-1);
    //----------------

}//Fin funcion setGraph
//--------------------------------
//Function that initializes values of R1,R2,R3 according to
//the Root vertex; and also define and initializes with 0s
//the temporal arrays
void setVariables()
{
    //Rs length = |NUMVERTICES|-1 because final path always
    //has one less than the #of vertices 
    R1source = (int *)calloc(NUMVERTICES-1,NUMVERTICES-1*sizeof(int));
    R2destination = (int *)calloc(NUMVERTICES-1,NUMVERTICES-1*sizeof(int));
    R3weigth = (int *)calloc(NUMVERTICES-1,NUMVERTICES-1*sizeof(int));

    //Set by default all the edges taking as the origin 
    //the root source, to all posible destinations
    int indxValidDestinations = -1;
    for(int i = 0; i < NUMVERTICES; i++)
    {

        //Only do not take as destination when source
        //and destination are equal
        if(C != i)
        {
            //Set source index
            R1source[indxValidDestinations] = C;

            indxValidDestinations++;
            R2destination[indxValidDestinations] = i;

            //Look for the actual weights in VE and VG
            //for the source and destination and in case
            //of not being found asign 0 as the weight
            int numDestinations = VG[i+NUMVERTICES];

            int startIndex = 0;
            for(int k = 0; k < i; k++)
            {
                startIndex = startIndex+VG[k+NUMVERTICES]
            }//End for

            //Recorrer solamente los destinos para el source
            for(int j = startIndex; j < numDestinations; j++)
            {
                int idDestino = VE[j];

                //Se encontro el destino .:. poner peso correspondiente
                if(idDestino == i)
                {
                    R3weigth[indxValidDestinations] = VE[j+NUMBEREDGES];
                }//End if

            }//End for 2

        }//End if


    }//Fin for 1


    //Define and construct T1 and T2? HERE
    //

}//End fucntions setVariables
//--------------------------------
void kernel1(int *v, int *e, int *r1, int * r2, int *r3, int c)
{
    //Define and construct T1 and T2
    T1 = (int *)calloc(MAZTRHEADSXBLOCK,MAZTRHEADSXBLOCK*sizeof(int));
    T2 = (int *)calloc(MAZTRHEADSXBLOCK,MAZTRHEADSXBLOCK*sizeof(int));

    //MIN REDUCTION AND WRITE RESULTS IN T1 AND T2

}//End ufnction kernel1
//--------------------------------
void primMST(int *v, int *e, int *r1, int * r2, int *r3, int c)
{
    //Define size of CUDA grid
    int g_row = (int)ceil((int)NUMBEREDGES/MAXTRHEADSXBLOCK);
    int g_col = (int)ceil((int)NUMBEREDGES/MAXTRHEADSXBLOCK); 
    dim3 bloques(g_col,g_row);
    dim3 hilos(MAXTRHEADSXBLOCK,MAXTRHEADSXBLOCK);

    hipEvent_t start, stop; 

    printf("Grid (%d,%d)\n", g_row, g_col); 

    //vARIABLES IN DEVICE
    int *VGD, *VED, *R1D, *R2D, R3D;
    int CD;

    //TRANSFER FROM HOST (CPU) TO DEVICE GPU
    hipSetDevice(gpudev);

    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start,0); 

    //1)Asignar memoria para variables en GPU
    hipMalloc(&VGD, NUMVERTICES*2*sizeof(int));
    hipMalloc(&VED, NUMBEREDGES*2*sizeof(int));
    hipMalloc(&R1D, NUMVERTICES-1*sizeof(int));
    hipMalloc(&R2D, NUMVERTICES-1*sizeof(int));
    hipMalloc(&R3D, NUMVERTICES-1*sizeof(int));
    hipMalloc(&R3D, NUMVERTICES-1*sizeof(int));
    hipMalloc(&CD, sizeof(int));

    //2)Copiar datos del host al device
    hipMemcpy(VGD,v,NUMVERTICES*2*sizeof(int),hipMemcpyDefault);
    hipMemcpy(VED,e,NUMBEREDGES*2*sizeof(int),hipMemcpyDefault);
    hipMemcpy(R1D,r1,NUMVERTICES-1*sizeof(int),hipMemcpyDefault);
    hipMemcpy(R2D,r2,NUMVERTICES-1*sizeof(int),hipMemcpyDefault);
    hipMemcpy(R3D,r3,NUMVERTICES-1*sizeof(int),hipMemcpyDefault);
    hipMemcpy(CD,c,sizeof(int),hipMemcpyDefault);

    //INICIO LOOP |NUMVERTICES|-1 VECES

    //3)Ejecutar kernel
    //INVOQUE KERNEL 1 AND WRITE RESULTS IN T1 AND T2
    kernel1<<bloques, hilos>>(VGD,VED,R1D,R2D,R3D,CD);

    //4)Copiar datos del device al host
    //T1 Y T2

    // Valores de T1[0] y T2[0] son añadidos
    // a los correspondientes R1 Y R3
    //T2[0] sobreescribe a C

    //FIN LOOP |NUMVERTICES|-1 VECES

    //5) Liberar Memoria
    hipFree(VGD);
    hipFree(VED);
    hipFree(R1D);
    hipFree(R2D);
    hipFree(R3D);
    hipFree(CD);

    hipEventRecord(stop, 0); 
    hipEventSynchronize(stop);

}//En function primMST
//---- FIN FUNCIONES -----
//Inicio del programa
int main(int argc, char **argv)
{
    setGraph();
    
    //Set root vertex of the MST
    C = 0;

    setVariables();

    primMST(VG,VE,R1,R2,R3,C);

    printf("Fin del programa V1\n");

}//Fin del main