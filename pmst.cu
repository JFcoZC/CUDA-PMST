
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdlib.h>

//VARIABLES GLOBALES
#define NUMVERTICES 5
#define MAXTRHEADSXBLOCK 32

//ID gpudevice that is used
int gpudev = 0;

//Graph representation with
int *EG;    //Double array of edges |NUMVERTICES|
int *VG;    //Double array of vertices

int C;      //Current vertex INDEX
int NUMBEREDGES;

//MST edge list: Shows the path that is followed.
int *R1source;
int *R2destination;
int *R3weigth;

//Temporal arrays used for reduction results
int *T1weights;
int *T2indexes;

//------- FUNCIONES --------
void printDoubleArray(int *VX)
{
    int lengthArray = NUMVERTICES;

    for(int i = 0; i <lengthArray; i++)
    {
        printf("%i ", VX[i]);

    }//End for 2

    printf("\n");

    for(int i = lengthArray; i < (lengthArray*2); i++)
    {
         printf("%i ",VX[i]);

    }//End for 2

    printf("\n");

}//Fin funcion printDoubleArray
//------------------------
void printArrayRange(int *VX,int start,int end)
{

    for(int i = start; i <= end; i++)
    {
        printf("%2i ", VX[i]);

    }//End for 2

    printf("\n");

}//Fin funcion printArrayRange
//------------------------
/*
*Function that creates the vaules of the graph
*in the strucutre.
*/
void setGraph()
{
    //Asign and initialize memory for the double array of integers
    //Because is a doble array it is multiplied by 2 only
    VG = (int *) calloc(NUMVERTICES*2, NUMVERTICES*2*sizeof(int) );
    
    int numberEdges = 0;
    int randValue = 0;

    //Inicializacion valores VG
    for (int i = 0; i < NUMVERTICES; i++)
    {
        //Set the index of the Vertex
        VG[i] = i;
        //Set in random way the # of vertices to 
        //wich this vertex is connected

        //#Of vertices can not be 0,becuase all the veritces
        //have to be connected so ensure that at least all
        // the nodes are connected to at leas 2 other vertices
        randValue = rand() % (NUMVERTICES-2) +2;
        VG[i+NUMVERTICES] = randValue;
        //Keep track of the number edges
        numberEdges = numberEdges + randValue;

    }//Fin for 1

    //!!!SAVE IN GLOBAL VARIABLE!!!
    NUMBEREDGES = numberEdges;

    //----------------
    printf("-- Source Vertex --\n");
    printDoubleArray(VG);
    printf("------\n");
    printf("TOTAL EDGES: %i\n",numberEdges);
    //----------------

    //Asign and initialize memory for the double array of integers
    //Because is a doble array it is multiplied by 2 only
    EG = (int *) calloc(numberEdges*2,numberEdges*2*sizeof(int));

    //Initialize EDGE Double array values
    int indxEdges = 0;

    for(int i = 0; i < NUMVERTICES; i++)
    {
        //Num of vertices to wich vertex i has a path
        int numVerticesConn = VG[i+NUMVERTICES];

        //1)Set the destinatio id of the vertex, which can not
        //be repeated and can not be the same as the source
        //vertex i
        int indxDestination = 0;

        //2)Set randomly the value of the weight of edge 1)
        //values of weight from 1 - 100 
        for(int j = 0; j < numVerticesConn; j++)
        {
            //1)
            //Ojo: indxDestination = j a menos que se encuentr
            //que source = destino; en ese caso y por el resto
            //del for j, indxDestination ira uno arriba que j
            if( i == j )
            {
                indxDestination++;
            }//End if

            EG[indxEdges] = indxDestination;

            //2)
            EG[indxEdges+numberEdges] = rand() % (100) +1;

            indxEdges++;
            indxDestination++; 

        }//End for 3

    }//Fin for 2

    //----------------
    printf("-- Destination vertex --\n");
    printArrayRange(EG,0,numberEdges-1);
    printf("-- Weigth of Edge --\n");
    printArrayRange(EG,numberEdges,(numberEdges*2)-1);
    //----------------

}//Fin funcion setGraph
//--------------------------------
//Function that initializes values of R1,R2,R3 according to
//the Root vertex; and also define and initializes with 0s
//the temporal arrays
void setVariables()
{
    //Rs length = |NUMVERTICES|-1 because final path always
    //has one less than the #of vertices 
    R1source = (int *)calloc(NUMVERTICES-1,NUMVERTICES-1*sizeof(int));
    R2destination = (int *)calloc(NUMVERTICES-1,NUMVERTICES-1*sizeof(int));
    R3weigth = (int *)calloc(NUMVERTICES-1,NUMVERTICES-1*sizeof(int));

    //Look for the actual weights in VE and VG
    //for the source and destination and in case
    //of not being found asign 0 as the weight
    int numDestinations = VG[C+NUMVERTICES];

    int startIndex = 0;
    for(int k = 0; k < C; k++)
    {
        startIndex = startIndex+VG[k+NUMVERTICES];
    }//End for

    numDestinations = numDestinations+startIndex;

    //----------
    //printf("Range of values in EG(%i - %i)\n", startIndex, numDestinations);
    //----------
    
    //Set by default all the edges taking as the origin 
    //the root source, to all posible destinations
    int indxValidDestinations = -1;
    for(int i = 0; i < NUMVERTICES; i++)
    {

        //Only do not take as destination when source
        //and destination are equal
        if(C != i)
        {
            indxValidDestinations++;

            //Set source index
            R1source[indxValidDestinations] = C;
            R2destination[indxValidDestinations] = i;

            //Recorrer solamente los destinos para el source
            for(int j = startIndex; j < numDestinations; j++)
            {
                int idDestino = EG[j];

                //Se encontro el destino .:. poner peso correspondiente
                //----------
                //printf("%i == %i\n", idDestino, i);
                //----------
                if(idDestino == i)
                {
                    R3weigth[indxValidDestinations] = EG[j+NUMBEREDGES];
                }//End if

            }//End for 2

        }//End if


    }//Fin for 1

    //--------------
    //Recordar que para el print se considera un elemento menos
    //del limite superior ya que realmnete hace el print hasta
    //la posicion indicada
    //printf("R1: \n");
    //printArrayRange(R1source,0,NUMVERTICES-2);
    //printf("R2: \n");
    //printArrayRange(R2destination,0,NUMVERTICES-2);
    //printf("R3: \n");
    //printArrayRange(R3weigth,0,NUMVERTICES-2);
    //--------------


}//End fucntions setVariables
//--------------------------------
__global__ void kernel1(int *v, int *e, int *r1, int * r2, int *r3, int *c, int *t1, int *t2)
{
    //Define and construct T1 and T2
    //T1weights = (int *)calloc(MAXTRHEADSXBLOCK,MAXTRHEADSXBLOCK*sizeof(int));
    //T2indexes = (int *)calloc(MAXTRHEADSXBLOCK,MAXTRHEADSXBLOCK*sizeof(int));

    int idBloque = blockIdx.x;
    //ID de cada hilo (IDHILOBLOQUE+IDBLOQUE*HILOSPORBLOQUE)
    int i = threadIdx.x + idBloque*blockDim.x;

    //MIN REDUCTION AND WRITE RESULTS IN T1 AND T2

    //1)All threads in the grid make reduction operation on an array
    //of input data, ann obtain min weight and index of each thread
    
    //Solo trabajar |v|-1 hilos 
    //V-1 porque Rs son de size |V|-1
    if( i < NUMVERTICES-1 )
    {
        printf("| idh: %i | ", i);
        printf("| %i : %i | ", r2[i], r3[i]);
        printf(" %i < %i //", r3[i*idBloque+i], t1[idBloque]);
        if(r3[i*idBloque+i] < t1[idBloque])
        {
            //Guardar Weight
            t1[idBloque] = r3[i*idBloque+i];

            //Guardar Indice
            t2[idBloque] = r2[i*idBloque+i];

        }//Nuevo menor encontrado

    }//End if 


    //2)All threads in every block make reduction of the result data in 1)
    //And obtain the minim value and index of every thread block

}//End ufnction kernel1
//--------------------------------
void primMST(int *v, int *e, int *r1, int * r2, int *r3, int c)
{
    //Define size of CUDA grid
    int g_row = (int)ceil((float)NUMVERTICES/(float)MAXTRHEADSXBLOCK);
    int g_col = (int)ceil((float)NUMVERTICES/(float)MAXTRHEADSXBLOCK); 
    int numBloques = g_row;
    dim3 bloques(g_col,g_row);
    dim3 hilos(MAXTRHEADSXBLOCK,MAXTRHEADSXBLOCK);

    hipEvent_t start, stop; 

    printf("Bloques: %i == %i \n", bloques, numBloques);
    printf("Hilos: %i \n", hilos);
    printf("Grid (%d,%d)\n", g_row, g_col); 

    //vARIABLES IN DEVICE
    int *VGD, *VED, *R1D, *R2D, *R3D;   //Arrays
    int *CD;                            //Variable 

    //Define and construct T1 and T2? HERE
    T1weights = (int *)calloc(numBloques,numBloques*sizeof(int));
    T2indexes = (int *)calloc(numBloques,numBloques*sizeof(int));

    //Initialize temporal weights with a very high value
    //in order to make that any wieght is better than 
    //the init value
    for(int i = 0; i < numBloques; i++)
    {
        T1weights[i] = 99999;

    }//End for 1

    //--------------
    //Recordar que para el print se considera un elemento menos
    //del limite superior ya que realmnete hace el print hasta
    //la posicion indicada
    printf("R1: \n");
    printArrayRange(r1,0,NUMVERTICES-2);
    printf("R2: \n");
    printArrayRange(r2,0,NUMVERTICES-2);
    printf("R3: \n");
    printArrayRange(r3,0,NUMVERTICES-2);
    //--------------

    //TRANSFER FROM HOST (CPU) TO DEVICE GPU
    hipSetDevice(gpudev);

    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start,0); 

    //1)Asignar memoria para variables en GPU
    hipMalloc(&VGD, NUMVERTICES*2*sizeof(int) );
    hipMalloc(&VED, NUMBEREDGES*2*sizeof(int) );
    hipMalloc(&R1D, (NUMVERTICES-1)*sizeof(int) );
    hipMalloc(&R2D, (NUMVERTICES-1)*sizeof(int) );
    hipMalloc(&R3D, (NUMVERTICES-1)*sizeof(int) );
    hipMalloc(&CD, int(sizeof(int)) );

    //2)Copiar datos del host al device
    hipMemcpy(VGD,v,NUMVERTICES*2*sizeof(int),hipMemcpyDefault);
    hipMemcpy(VED,e,NUMBEREDGES*2*sizeof(int),hipMemcpyDefault);
    hipMemcpy(R1D,r1,NUMVERTICES-1*sizeof(int),hipMemcpyDefault);
    hipMemcpy(R2D,r2,NUMVERTICES-1*sizeof(int),hipMemcpyDefault);
    hipMemcpy(R3D,r3,NUMVERTICES-1*sizeof(int),hipMemcpyDefault);
    hipMemcpy(CD,&c,sizeof(int),hipMemcpyDefault);

    //INICIO LOOP |NUMVERTICES|-1 VECES

    //3)Ejecutar kernel
    //INVOQUE KERNEL 1 AND WRITE RESULTS IN T1 AND T2
    kernel1<<<bloques, hilos>>>(VGD,VED,R1D,R2D,R3D,CD,T1weights,T2indexes);

    //4)Copiar datos del device al host
    //T1 Y T2
    

    //Verificar si se invica a Kernel 2
    if(numBloques > 1)
    {
        printf("Invoke Kernel2\n");

    }//End if

    // Valores de T1[0] y T2[0] son añadidos
    // a los correspondientes R1 Y R3
    //T2[0] sobreescribe a C
    //---------------
    printf("Minimum weight found: %i for vertex with ID: %i \n", T1weights[0], T2indexes[0]);
    //---------------

    //FIN LOOP |NUMVERTICES|-1 VECES

    //5) Liberar Memoria
    hipFree(VGD);
    hipFree(VED);
    hipFree(R1D);
    hipFree(R2D);
    hipFree(R3D);
    hipFree(CD);

    hipEventRecord(stop, 0); 
    hipEventSynchronize(stop);

}//En function primMST
//---- FIN FUNCIONES -----
//Inicio del programa
int main(int argc, char **argv)
{
    setGraph();
    
    //Set root vertex of the MST
    C = 4;

    setVariables();

    printf("IDs threads: \n");
    primMST(VG,EG,R1source,R2destination,R3weigth,C);
    printf("\n");

    //---------------
    printf("Minimum weight found: %i for vertex with ID: %i \n", T1weights[0], T2indexes[0]);
    //---------------

    printf("Fin del programa V1\n");

}//Fin del main