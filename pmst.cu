
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdlib.h>

//NUMERO BLOQUES = INTEGERMASALTO(NUMVERTICES/MAXTRHEADSXBLOCK)
//VARIABLES GLOBALES
#define NUMVERTICES 300      //MAXIMUM 300
#define MAXTRHEADSXBLOCK 32 //[1 - 32]

//ID gpudevice that is used
int gpudev = 0;

//Graph representation with
int *EG;    //Double array of edges |NUMVERTICES|
int *VG;    //Double array of vertices

int C;      //Current vertex INDEX
int NUMBEREDGES;

//MST edge list: Shows the path that is followed.
int *R1source;
int *R2destination;
int *R3weigth;

//Temporal arrays used for reduction results
int *T1weights;
int *T2indexes;

//------- FUNCIONES --------
void printDoubleArray(int *VX)
{
    int lengthArray = NUMVERTICES;

    for(int i = 0; i <lengthArray; i++)
    {
        printf("%i ", VX[i]);

    }//End for 2

    printf("\n");

    for(int i = lengthArray; i < (lengthArray*2); i++)
    {
         printf("%i ",VX[i]);

    }//End for 2

    printf("\n");

}//Fin funcion printDoubleArray
//------------------------
void printArrayRange(int *VX,int start,int end)
{

    for(int i = start; i <= end; i++)
    {
        printf("%2i ", VX[i]);

    }//End for 2

    printf("\n");

}//Fin funcion printArrayRange
//------------------------
/*
*Function that creates the vaules of the graph
*in the strucutre.
*/
void setGraph()
{
    //Asign and initialize memory for the double array of integers
    //Because is a doble array it is multiplied by 2 only
    VG = (int *) calloc(NUMVERTICES*2, NUMVERTICES*2*sizeof(int) );
    
    int numberEdges = 0;
    int randValue = 0;

    //Inicializacion valores VG
    for (int i = 0; i < NUMVERTICES; i++)
    {
        //Set the index of the Vertex
        VG[i] = i;
        //Set in random way the # of vertices to 
        //wich this vertex is connected

        //#Of vertices can not be 0,becuase all the veritces
        //have to be connected so ensure that at least all
        // the nodes are connected to at leas 2 other vertices
        randValue = rand() % (NUMVERTICES-2) +2;
        VG[i+NUMVERTICES] = randValue;
        //Keep track of the number edges
        numberEdges = numberEdges + randValue;

    }//Fin for 1

    //!!!SAVE IN GLOBAL VARIABLE!!!
    NUMBEREDGES = numberEdges;

    //----------------
    printf("-- Source Vertex --\n");
    printDoubleArray(VG);
    printf("------\n");
    printf("TOTAL EDGES: %i\n",numberEdges);
    //----------------

    //Asign and initialize memory for the double array of integers
    //Because is a doble array it is multiplied by 2 only
    EG = (int *) calloc(numberEdges*2,numberEdges*2*sizeof(int));

    //Initialize EDGE Double array values
    int indxEdges = 0;

    for(int i = 0; i < NUMVERTICES; i++)
    {
        //Num of vertices to wich vertex i has a path
        int numVerticesConn = VG[i+NUMVERTICES];

        //1)Set the destinatio id of the vertex, which can not
        //be repeated and can not be the same as the source
        //vertex i
        int indxDestination = 0;

        //2)Set randomly the value of the weight of edge 1)
        //values of weight from 1 - 100 
        for(int j = 0; j < numVerticesConn; j++)
        {
            //1)
            //Ojo: indxDestination = j a menos que se encuentr
            //que source = destino; en ese caso y por el resto
            //del for j, indxDestination ira uno arriba que j
            if( i == j )
            {
                indxDestination++;
            }//End if

            EG[indxEdges] = indxDestination;

            //2)
            EG[indxEdges+numberEdges] = rand() % (100) +1;

            indxEdges++;
            indxDestination++; 

        }//End for 3

    }//Fin for 2

    //----------------
    //printf("-- Destination vertex --\n");
    //printArrayRange(EG,0,numberEdges-1);
    //printf("-- Weigth of Edge --\n");
    //printArrayRange(EG,numberEdges,(numberEdges*2)-1);
    //----------------

}//Fin funcion setGraph
//--------------------------------
//Function that initializes values of R1,R2,R3 according to
//the Root vertex; and also define and initializes with 0s
//the temporal arrays
void setVariables()
{
    //Rs length = |NUMVERTICES|-1 because final path always
    //has one less than the #of vertices 
    R1source = (int *)calloc(NUMVERTICES-1,NUMVERTICES-1*sizeof(int));
    R2destination = (int *)calloc(NUMVERTICES-1,NUMVERTICES-1*sizeof(int));
    R3weigth = (int *)calloc(NUMVERTICES-1,NUMVERTICES-1*sizeof(int));

    //Look for the actual weights in VE and VG
    //for the source and destination and in case
    //of not being found asign 0 as the weight
    int numDestinations = VG[C+NUMVERTICES];

    int startIndex = 0;
    for(int k = 0; k < C; k++)
    {
        startIndex = startIndex+VG[k+NUMVERTICES];
    }//End for

    numDestinations = numDestinations+startIndex;

    //----------
    //printf("Range of values in EG(%i - %i)\n", startIndex, numDestinations);
    //----------
    
    //Set by default all the edges taking as the origin 
    //the root source, to all posible destinations
    int indxValidDestinations = -1;
    for(int i = 0; i < NUMVERTICES; i++)
    {

        //Only do not take as destination when source
        //and destination are equal
        if(C != i)
        {
            indxValidDestinations++;

            //Set source index
            R1source[indxValidDestinations] = C;
            R2destination[indxValidDestinations] = i;

            //Recorrer solamente los destinos para el source
            for(int j = startIndex; j < numDestinations; j++)
            {
                int idDestino = EG[j];

                //Se encontro el destino .:. poner peso correspondiente
                //----------
                //printf("%i == %i\n", idDestino, i);
                //----------
                if(idDestino == i)
                {
                    R3weigth[indxValidDestinations] = EG[j+NUMBEREDGES];
                }//End if

            }//End for 2

        }//End if


    }//Fin for 1

    //--------------
    //Recordar que para el print se considera un elemento menos
    //del limite superior ya que realmnete hace el print hasta
    //la posicion indicada
    //printf("R1: \n");
    //printArrayRange(R1source,0,NUMVERTICES-2);
    //printf("R2: \n");
    //printArrayRange(R2destination,0,NUMVERTICES-2);
    //printf("R3: \n");
    //printArrayRange(R3weigth,0,NUMVERTICES-2);
    //--------------


}//End fucntions setVariables
//--------------------------------
__global__ void kernel1(int *v, int *e, int *r1, int *r2, int *r3, int *c, int *t1, int *t2)
{
    //Define and construct T1 and T2
    //T1weights = (int *)calloc(MAXTRHEADSXBLOCK,MAXTRHEADSXBLOCK*sizeof(int));
    //T2indexes = (int *)calloc(MAXTRHEADSXBLOCK,MAXTRHEADSXBLOCK*sizeof(int));

    int idBloque = blockIdx.x;
    //ID de cada hilo (IDHILOBLOQUE+IDBLOQUE*HILOSPORBLOQUE)
    int i = threadIdx.x + idBloque*blockDim.x;

    //MIN REDUCTION AND WRITE RESULTS IN T1 AND T2

    //1)All threads in the grid make reduction operation on an array
    //of input data, ann obtain min weight and index of each thread
    
    //Solo trabajar |v|-1 hilos 
    //V-1 porque Rs son de size |V|-1
    if( i < NUMVERTICES-1 )
    {
        //----------------------
        //printf("| idh: %i | ", i);
        //printf("| %i %i | ", v[i],v[i+NUMVERTICES]);
        //----------------------

        //---------------------
        //printf("| %i  %i : %i | ", r1[i], r2[2], r3[2]);
        //printf(" %i < %i //", r3[i], t1[idBloque]);
        //---------------------
        //Con weiht mwnor al actual pero que sea un
        //weigth valido (diferente de 0)
        if(r3[i] < t1[idBloque] && r3[i] != 0 )
        {
            //Guardar Weight
            t1[idBloque] = r3[i];

            //Guardar Indice
            t2[idBloque] = r2[i];

        }//Nuevo menor encontrado

    }//End if

    //printf("| %i | ", r3[i]);

    //i < MAXNUMBEREDGES
    /*if(i < 15)
    {
                                 //[i+MAXNUMBEREDGES]
        printf("! %i %i ! ", e[i],e[i+15]);

    }//End if*/


    //2)All threads in every block make reduction of the result data in 1)
    //And obtain the minim value and index of every thread block

}//End function kernel1
//--------------------------------
__global__ void kernel2(int *numBlocks, int *weights, int *indxs)
{
    int N = numBlocks[0];

    //Reservar espacio en zona de memoria compartida
    __shared__ int temporal[MAXTRHEADSXBLOCK];

    //Indice de cada hilo en un solo bloque
    int i = threadIdx.x;
    
    if(i < N)
    {
        //Copiamos el vector de pesos en temporal y sincronizamos
        temporal[i] = weights[i];
        __syncthreads();

        //---------------------
        printf("|%i)  %i : %i | ", i ,weights[i], indxs[i]);
        //printf("| %i | ", weights[0]);
        //----------------------

    }//End if


}//End function kernel2
//--------------------------------
void primMST(int *v, int *e, int *r1, int * r2, int *r3, int c)
{
    //Define size of CUDA grid
    int g_row = (int)ceil((float)NUMVERTICES/(float)MAXTRHEADSXBLOCK);
    int g_col = (int)ceil((float)NUMVERTICES/(float)MAXTRHEADSXBLOCK); 
    int numBloques = g_row;
    dim3 bloques(g_col,g_row);
    dim3 hilos(MAXTRHEADSXBLOCK,MAXTRHEADSXBLOCK);

    hipEvent_t start, stop; 

    printf("Bloques: %i == %i \n", bloques, numBloques);
    printf("Hilos: %i \n", hilos);
    printf("Grid (%d,%d)\n", g_row, g_col); 

    //vARIABLES IN DEVICE
    int *VGD, *VED, *R1D, *R2D, *R3D;   //Arrays
    int *T1D, *T2D;
    int *CD;                            //Variable 

    //Define and construct T1 and T2? HERE
    T1weights = (int *)calloc(numBloques,numBloques*sizeof(int));
    T2indexes = (int *)calloc(numBloques,numBloques*sizeof(int));

    //Initialize temporal weights with a very high value
    //in order to make that any wieght is better than 
    //the init value
    for(int i = 0; i < numBloques; i++)
    {
        T1weights[i] = 99999;

    }//End for 1

    //--------------
    //Recordar que para el print se considera un elemento menos
    //del limite superior ya que realmnete hace el print hasta
    //la posicion indicada
    printf("R1: \n");
    printArrayRange(r1,0,NUMVERTICES-2);
    printf("R2: \n");
    printArrayRange(r2,0,NUMVERTICES-2);
    printf("R3: \n");
    printArrayRange(r3,0,NUMVERTICES-2);
    //--------------

    //TRANSFER FROM HOST (CPU) TO DEVICE GPU
    hipSetDevice(gpudev);

    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start,0); 

    //1)Asignar memoria para variables en GPU
    hipMalloc(&VGD, NUMVERTICES*2*sizeof(int) );
    hipMalloc(&VED, NUMBEREDGES*2*sizeof(int) );
    hipMalloc(&R1D, (NUMVERTICES-1)*sizeof(int) );
    hipMalloc(&R2D, (NUMVERTICES-1)*sizeof(int) );
    hipMalloc(&R3D, (NUMVERTICES-1)*sizeof(int) );
    hipMalloc(&T1D, (numBloques)*sizeof(int) );
    hipMalloc(&T2D, (numBloques)*sizeof(int) );
    hipMalloc(&CD, int(sizeof(int)) );

    //2)Copiar datos del host al device
    hipMemcpy(VGD,v,NUMVERTICES*2*sizeof(int),hipMemcpyDefault);
    hipMemcpy(VED,e,NUMBEREDGES*2*sizeof(int),hipMemcpyDefault);
    hipMemcpy(R1D,r1,(NUMVERTICES-1)*sizeof(int),hipMemcpyDefault);
    hipMemcpy(R2D,r2,(NUMVERTICES-1)*sizeof(int),hipMemcpyDefault);
    hipMemcpy(R3D,r3,(NUMVERTICES-1)*sizeof(int),hipMemcpyDefault);
    hipMemcpy(T1D,T1weights,numBloques*sizeof(int),hipMemcpyDefault);
    hipMemcpy(T2D,T2indexes,numBloques*sizeof(int),hipMemcpyDefault);
    hipMemcpy(CD,&c,sizeof(int),hipMemcpyDefault);

    //INICIO LOOP |NUMVERTICES|-1 VECES

    //3)Ejecutar kernel
    //INVOQUE KERNEL 1 AND WRITE RESULTS IN T1 AND T2
    kernel1<<<bloques, hilos>>>(VGD,VED,R1D,R2D,R3D,CD,T1D,T2D);

    //4)Copiar datos del device al host
    //T1 Y T2
    
    // Valores de T1[0] y T2[0] son añadidos
    // a los correspondientes R1 Y R3
    //T2[0] sobreescribe a C
    hipMemcpy(T1weights,T1D,numBloques*sizeof(int),hipMemcpyDefault);
    hipMemcpy(T2indexes,T2D,numBloques*sizeof(int),hipMemcpyDefault);
    //---------------
    printf("\n Minimum weight found for each block \n");
    printf("Id: \n");
    printArrayRange(T2indexes,0,numBloques-1);
    printf("Weight: \n");
    printArrayRange(T1weights,0,numBloques-1);
    //---------------

    //Verificar si se inicia al Kernel 2
    //MAXTRHEADSXBLOCK > numBloques > 1
    if(numBloques > 1)
    {
        //Definir variable en device
        int *NBD;

        //1)Asinar memoria para vairable en GPU/device
        hipMalloc(&NBD, int(sizeof(int)) );

        //2)Copiar datos del host al device
        //cudaMemcpy(T1D,T1weights,numBloques*sizeof(int),cudaMemcpyDefault);
        //cudaMemcpy(T2D,T2indexes,numBloques*sizeof(int),cudaMemcpyDefault);
        hipMemcpy(NBD,&numBloques,sizeof(int),hipMemcpyDefault);

        //3)ejecutar kermel2
        printf("Invoke Kernel2\n");
        kernel2<<<1,hilos>>>(NBD,T1D,T2D);

        //4)Copiar datos del device al host
        hipMemcpy(T1weights,T1D,numBloques*sizeof(int),hipMemcpyDefault);
        hipMemcpy(T2indexes,T2D,numBloques*sizeof(int),hipMemcpyDefault);

        //5)liberar memoria

    }//End if

    //---------------
    printf("Minimum weight found: %i for vertex with ID: %i \n", T1weights[0], T2indexes[0]);
    //---------------

    //FIN LOOP |NUMVERTICES|-1 VECES


    //5) Liberar Memoria
    hipFree(VGD);
    hipFree(VED);
    hipFree(R1D);
    hipFree(R2D);
    hipFree(R3D);
    hipFree(T1D);
    hipFree(T2D);
    hipFree(CD);

    hipEventRecord(stop, 0); 
    hipEventSynchronize(stop);

}//En function primMST
//---- FIN FUNCIONES -----
//Inicio del programa
int main(int argc, char **argv)
{
    setGraph();
    
    //Set root vertex of the MST
    C = 0;

    setVariables();

    printf("IDs threads: \n");
    primMST(VG,EG,R1source,R2destination,R3weigth,C);
    printf("\n");


    printf("Fin del programa V3\n");

}//Fin del main