
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdlib.h>

//NUMERO BLOQUES = INTEGERMASALTO(NUMVERTICES/MAXTRHEADSXBLOCK)
//VARIABLES GLOBALES
#define NUMVERTICES 300      //MAXIMUM 300
#define MAXTRHEADSXBLOCK 32 //[1 - 32]

//ID gpudevice that is used
int gpudev = 0;

//Graph representation with
int *EG;    //Double array of edges |NUMVERTICES|
int *VG;    //Double array of vertices

int C;      //Current vertex INDEX
int NUMBEREDGES;

//MST edge list: Shows the path that is followed.
int *R1source;
int *R2destination;
int *R3weigth;

//Temporal arrays used for reduction results
int *T1weights;
int *T2indexes;

//------- FUNCIONES --------
void printDoubleArray(int *VX)
{
    int lengthArray = NUMVERTICES;

    for(int i = 0; i <lengthArray; i++)
    {
        printf("%i ", VX[i]);

    }//End for 2

    printf("\n");

    for(int i = lengthArray; i < (lengthArray*2); i++)
    {
         printf("%i ",VX[i]);

    }//End for 2

    printf("\n");

}//Fin funcion printDoubleArray
//------------------------
void printArrayRange(int *VX,int start,int end)
{

    for(int i = start; i <= end; i++)
    {
        printf("%2i ", VX[i]);

    }//End for 2

    printf("\n");

}//Fin funcion printArrayRange
//------------------------
/*
*Function that creates the vaules of the graph
*in the strucutre.
*/
void setGraph()
{
    //Asign and initialize memory for the double array of integers
    //Because is a doble array it is multiplied by 2 only
    VG = (int *) calloc(NUMVERTICES*2, NUMVERTICES*2*sizeof(int) );
    
    int numberEdges = 0;
    int randValue = 0;

    //Inicializacion valores VG
    for (int i = 0; i < NUMVERTICES; i++)
    {
        //Set the index of the Vertex
        VG[i] = i;
        //Set in random way the # of vertices to 
        //wich this vertex is connected

        //#Of vertices can not be 0,becuase all the veritces
        //have to be connected so ensure that at least all
        // the nodes are connected to at leas 2 other vertices
        randValue = rand() % (NUMVERTICES-2) +2;
        VG[i+NUMVERTICES] = randValue;
        //Keep track of the number edges
        numberEdges = numberEdges + randValue;

    }//Fin for 1

    //!!!SAVE IN GLOBAL VARIABLE!!!
    NUMBEREDGES = numberEdges;

    //----------------
    printf("-- Source Vertex --\n");
    printDoubleArray(VG);
    printf("------\n");
    printf("TOTAL EDGES: %i\n",numberEdges);
    //----------------

    //Asign and initialize memory for the double array of integers
    //Because is a doble array it is multiplied by 2 only
    EG = (int *) calloc(numberEdges*2,numberEdges*2*sizeof(int));

    //Initialize EDGE Double array values
    int indxEdges = 0;

    for(int i = 0; i < NUMVERTICES; i++)
    {
        //Num of vertices to wich vertex i has a path
        int numVerticesConn = VG[i+NUMVERTICES];

        //1)Set the destinatio id of the vertex, which can not
        //be repeated and can not be the same as the source
        //vertex i
        int indxDestination = 0;

        //2)Set randomly the value of the weight of edge 1)
        //values of weight from 1 - 100 
        for(int j = 0; j < numVerticesConn; j++)
        {
            //1)
            //Ojo: indxDestination = j a menos que se encuentr
            //que source = destino; en ese caso y por el resto
            //del for j, indxDestination ira uno arriba que j
            if( i == j )
            {
                indxDestination++;
            }//End if

            EG[indxEdges] = indxDestination;

            //2)
            EG[indxEdges+numberEdges] = rand() % (100) +1;

            indxEdges++;
            indxDestination++; 

        }//End for 3

    }//Fin for 2

    //----------------
    //printf("-- Destination vertex --\n");
    //printArrayRange(EG,0,numberEdges-1);
    //printf("-- Weigth of Edge --\n");
    //printArrayRange(EG,numberEdges,(numberEdges*2)-1);
    //----------------

}//Fin funcion setGraph
//--------------------------------
//Function that initializes values of R1,R2,R3 according to
//the Root vertex; and also define and initializes with 0s
//the temporal arrays
void setVariables()
{
    //Rs length = |NUMVERTICES|-1 because final path always
    //has one less than the #of vertices 
    R1source = (int *)calloc(NUMVERTICES-1,NUMVERTICES-1*sizeof(int));
    R2destination = (int *)calloc(NUMVERTICES-1,NUMVERTICES-1*sizeof(int));
    R3weigth = (int *)calloc(NUMVERTICES-1,NUMVERTICES-1*sizeof(int));

    //Look for the actual weights in VE and VG
    //for the source and destination and in case
    //of not being found asign 0 as the weight
    int numDestinations = VG[C+NUMVERTICES];

    int startIndex = 0;
    for(int k = 0; k < C; k++)
    {
        startIndex = startIndex+VG[k+NUMVERTICES];
    }//End for

    numDestinations = numDestinations+startIndex;

    //----------
    //printf("Range of values in EG(%i - %i)\n", startIndex, numDestinations);
    //----------
    
    //Set by default all the edges taking as the origin 
    //the root source, to all posible destinations
    int indxValidDestinations = -1;
    for(int i = 0; i < NUMVERTICES; i++)
    {

        //Only do not take as destination when source
        //and destination are equal
        if(C != i)
        {
            indxValidDestinations++;

            //Set source index
            R1source[indxValidDestinations] = C;
            R2destination[indxValidDestinations] = i;

            //Recorrer solamente los destinos para el source
            for(int j = startIndex; j < numDestinations; j++)
            {
                int idDestino = EG[j];

                //Se encontro el destino .:. poner peso correspondiente
                //----------
                //printf("%i == %i\n", idDestino, i);
                //----------
                if(idDestino == i)
                {
                    R3weigth[indxValidDestinations] = EG[j+NUMBEREDGES];
                }//End if

            }//End for 2

        }//End if


    }//Fin for 1

    //--------------
    //Recordar que para el print se considera un elemento menos
    //del limite superior ya que realmnete hace el print hasta
    //la posicion indicada
    //printf("R1: \n");
    //printArrayRange(R1source,0,NUMVERTICES-2);
    //printf("R2: \n");
    //printArrayRange(R2destination,0,NUMVERTICES-2);
    //printf("R3: \n");
    //printArrayRange(R3weigth,0,NUMVERTICES-2);
    //--------------


}//End fucntions setVariables
//--------------------------------
__global__ void kernel1(int *v, int *e, int *r1, int *r2, int *r3, int *c, int *t1, int *t2)
{
    //Define and construct T1 and T2
    //T1weights = (int *)calloc(MAXTRHEADSXBLOCK,MAXTRHEADSXBLOCK*sizeof(int));
    //T2indexes = (int *)calloc(MAXTRHEADSXBLOCK,MAXTRHEADSXBLOCK*sizeof(int));

    int idBloque = blockIdx.x;
    //ID de cada hilo (IDHILOBLOQUE+IDBLOQUE*HILOSPORBLOQUE)
    int i = threadIdx.x + idBloque*blockDim.x;

    //MIN REDUCTION AND WRITE RESULTS IN T1 AND T2

    //1)All threads in the grid make reduction operation on an array
    //of input data, ann obtain min weight and index of each thread
    
    //Solo trabajar |v|-1 hilos 
    //V-1 porque Rs son de size |V|-1
    if( i < NUMVERTICES-1 )
    {
        //----------------------
        //printf("| idh: %i | ", i);
        //printf("| %i %i | ", v[i],v[i+NUMVERTICES]);
        //----------------------

        //---------------------
        //printf("| %i  %i : %i | ", r1[i], r2[2], r3[2]);
        //printf(" %i < %i //", r3[i], t1[idBloque]);
        //---------------------
        //Con weiht mwnor al actual pero que sea un
        //weigth valido (diferente de 0)
        if(r3[i] < t1[idBloque] && r3[i] != 0 )
        {
            //Guardar Weight
            t1[idBloque] = r3[i];

            //Guardar Indice
            t2[idBloque] = r2[i];

        }//Nuevo menor encontrado

    }//End if

    //printf("| %i | ", r3[i]);

    //i < MAXNUMBEREDGES
    /*if(i < 15)
    {
                                 //[i+MAXNUMBEREDGES]
        printf("! %i %i ! ", e[i],e[i+15]);

    }//End if*/


    //2)All threads in every block make reduction of the result data in 1)
    //And obtain the minim value and index of every thread block

}//End function kernel1
//--------------------------------
__global__ void kernel2(int *numBlocks, int *weights, int *indxs)
{
    int N = numBlocks[0];

    //Reservar espacio en zona de memoria compartida
    __shared__ int temporal[MAXTRHEADSXBLOCK];
    __shared__ int tempids[MAXTRHEADSXBLOCK];

    //Indice de cada hilo en un solo bloque
    int i = threadIdx.x;
    
    if(i < N)
    {
        //Copiamos el vector de pesos en temporal y sincronizamos
        temporal[i] = weights[i];
        tempids[i] = indxs[i];
        __syncthreads();

        //---------------------
        //printf("|%i)  %i : %i | ", i ,weights[i], indxs[i]);
        //printf("| %i | ", temporal[i]);
        //----------------------

        //Inicio de reduccion paralela
        int salto = N/2;

        //log2(N) iteraciones
        while(salto)
        {
            //Solo trabajan la mitad de los hilos
            if(i < salto)
            {
                //Si se encuentra un vertex con peso menor se elige
                //como mejor candidato
                if( temporal[i+salto] < temporal[i] && temporal[i] != 0 )
                {
                    temporal[i] = temporal[i+salto];
                    tempids[i] = tempids[i+salto];

                }//End if

            }//End if 2
            __syncthreads();
            salto = salto/2;

        }//End while

        //Hilo 0 escibe el resultado final en memoria global
        if(i == 0)
        {
            weights[0] = temporal[0];
            indxs[0] = tempids[0];

        }//End if 2

    }//End if 1 


}//End function kernel2
//--------------------------------
//Comparing and update MST
__global__ void kernel3(int *v, int *e, int *r1, int *r3, int *c, int *numEdges)
{
    //1)Read current Vertex index

    //2)Fin the weight between current vertex and the
    //other vertices (n other Vertex in actual moment)

    //For every Vertex n if new weight with this C vertex is < old weight
    // Adjust corresponding values of R1 and R3 by :
    //if(W[n] < R3[n] )
    //R1[n] = C
    //R3[n] = W[n]

    //Look for the actual weights in VE and VG
    //for the source and destination and in case
    //of not being found asign 0 as the weight
    int NE = numEdges[0];
    int C = c[0];
    int numDestinations = v[C+NUMVERTICES];

    int startIndex = 0;
    for(int k = 0; k < C; k++)
    {
        startIndex = startIndex+v[k+NUMVERTICES];
    }//End for

    numDestinations = numDestinations+startIndex;

    //----------
    //printf("Range of values in EG(%i - %i)\n", startIndex, numDestinations);
    //----------

    int idBloque = blockIdx.x;
    //ID de cada hilo (IDHILOBLOQUE+IDBLOQUE*HILOSPORBLOQUE)
    int i = threadIdx.x + idBloque*blockDim.x;
    
    //Set by default all the edges taking as the origin 
    //the root source, to all posible destinations
    int indxValidDestinations = -1;
    if(i < NUMVERTICES)
    {

        //Only do not take as destination when source
        //and destination are equal
        if(C != i)
        {
            indxValidDestinations++;

            //Recorrer solamente los destinos para el source
            for(int j = startIndex; j < numDestinations; j++)
            {
                int idDestino = e[j];

                //Se encontro el destino .:. poner peso correspondiente
                //----------
                //printf("%i == %i\n", idDestino, i);
                //----------
                if(idDestino == i)
                {

                    if(e[j+NE] < r3[indxValidDestinations])
                    {
                        r3[indxValidDestinations] = e[j+NE];
                        r1[indxValidDestinations] = C;

                    }//End if

                }//End if

            }//End for 2

        }//End if


    }//Fin for 1
    

}//End function kernel3
//--------------------------------
void primMST(int *v, int *e, int *r1, int * r2, int *r3, int c)
{
    //Define size of CUDA grid
    int g_row = (int)ceil((float)NUMVERTICES/(float)MAXTRHEADSXBLOCK);
    int g_col = (int)ceil((float)NUMVERTICES/(float)MAXTRHEADSXBLOCK); 
    int numBloques = g_row;
    dim3 bloques(g_col,g_row);
    dim3 hilos(MAXTRHEADSXBLOCK,MAXTRHEADSXBLOCK);

    hipEvent_t start, stop; 

    printf("Bloques: %i == %i \n", bloques, numBloques);
    printf("Hilos: %i \n", hilos);
    printf("Grid (%d,%d)\n", g_row, g_col); 

    //vARIABLES IN DEVICE
    int *VGD, *VED, *R1D, *R2D, *R3D;   //Arrays
    int *T1D, *T2D;
    int *CD, *NED;                      //Variables 

    //Define and construct T1 and T2? HERE
    T1weights = (int *)calloc(numBloques,numBloques*sizeof(int));
    T2indexes = (int *)calloc(numBloques,numBloques*sizeof(int));

    //Initialize temporal weights with a very high value
    //in order to make that any wieght is better than 
    //the init value
    for(int i = 0; i < numBloques; i++)
    {
        T1weights[i] = 99999;

    }//End for 1

    //--------------
    //Recordar que para el print se considera un elemento menos
    //del limite superior ya que realmnete hace el print hasta
    //la posicion indicada
    printf("R1: \n");
    printArrayRange(r1,0,NUMVERTICES-2);
    printf("R2: \n");
    printArrayRange(r2,0,NUMVERTICES-2);
    printf("R3: \n");
    printArrayRange(r3,0,NUMVERTICES-2);
    //--------------

    //TRANSFER FROM HOST (CPU) TO DEVICE GPU
    hipSetDevice(gpudev);

    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start,0); 

    //1)Asignar memoria para variables en GPU
    hipMalloc(&VGD, NUMVERTICES*2*sizeof(int) );
    hipMalloc(&VED, NUMBEREDGES*2*sizeof(int) );
    hipMalloc(&R1D, (NUMVERTICES-1)*sizeof(int) );
    hipMalloc(&R2D, (NUMVERTICES-1)*sizeof(int) );
    hipMalloc(&R3D, (NUMVERTICES-1)*sizeof(int) );
    hipMalloc(&T1D, (numBloques)*sizeof(int) );
    hipMalloc(&T2D, (numBloques)*sizeof(int) );
    hipMalloc(&CD, int(sizeof(int)) );
    hipMalloc(&NED, int(sizeof(int)) );


    //INICIO LOOP |NUMVERTICES|-1 VECES
    int iCountVer = 0;

    while(iCountVer < NUMVERTICES-1)
    {
        //----
        printf("---- %i) ----- \n", iCountVer);
        printf("Current vertex: %i \n", c);
        //----

        //2)Copiar datos del host al device
        hipMemcpy(VGD,v,NUMVERTICES*2*sizeof(int),hipMemcpyDefault);
        hipMemcpy(VED,e,NUMBEREDGES*2*sizeof(int),hipMemcpyDefault);
        hipMemcpy(R1D,r1,(NUMVERTICES-1)*sizeof(int),hipMemcpyDefault);
        hipMemcpy(R2D,r2,(NUMVERTICES-1)*sizeof(int),hipMemcpyDefault);
        hipMemcpy(R3D,r3,(NUMVERTICES-1)*sizeof(int),hipMemcpyDefault);
        hipMemcpy(T1D,T1weights,numBloques*sizeof(int),hipMemcpyDefault);
        hipMemcpy(T2D,T2indexes,numBloques*sizeof(int),hipMemcpyDefault);
        hipMemcpy(CD,&c,sizeof(int),hipMemcpyDefault);
        hipMemcpy(NED,&NUMBEREDGES,sizeof(int),hipMemcpyDefault);

        //3)Ejecutar kernel
        //INVOQUE KERNEL 1 AND WRITE RESULTS IN T1 AND T2
        kernel1<<<bloques, hilos>>>(VGD,VED,R1D,R2D,R3D,CD,T1D,T2D);

        //4)Copiar datos del device al host
        //T1 Y T2
        
        // Valores de T1[0] y T2[0] son añadidos
        // a los correspondientes R1 Y R3
        //T2[0] sobreescribe a C
        hipMemcpy(T1weights,T1D,numBloques*sizeof(int),hipMemcpyDefault);
        hipMemcpy(T2indexes,T2D,numBloques*sizeof(int),hipMemcpyDefault);
        //---------------
        printf("\n Minimum weight found for each block (Global memory reduction) \n");
        printf("Id: \n");
        printArrayRange(T2indexes,0,numBloques-1);
        printf("Weight: \n");
        printArrayRange(T1weights,0,numBloques-1);
        //---------------

        //Verificar si se inicia al Kernel 2
        //MAXTRHEADSXBLOCK > numBloques > 1
        if(numBloques > 1)
        {
            //Definir variable en device
            int *NBD;

            //1)Asinar memoria para vairable en GPU/device
            hipMalloc(&NBD, int(sizeof(int)) );

            //2)Copiar datos del host al device
            hipMemcpy(NBD,&numBloques,sizeof(int),hipMemcpyDefault);

            //3)ejecutar kermel2
            printf("Invoke Kernel2\n");
            kernel2<<<1,hilos>>>(NBD,T1D,T2D);

            //4)Copiar datos del device al host
            hipMemcpy(T1weights,T1D,numBloques*sizeof(int),hipMemcpyDefault);
            hipMemcpy(T2indexes,T2D,numBloques*sizeof(int),hipMemcpyDefault);

            //---------------
            printf("\n 2) Minimum weight found of each block (After shared memory reduction) \n");
            printf("Id: \n");
            printArrayRange(T2indexes,0,numBloques-1);
            printf("Weight: \n");
            printArrayRange(T1weights,0,numBloques-1);
            //---------------
            
            //5)liberar memoria NBD

        }//End if kernel2

        //---------------
        printf("Minimum weight found: %i for vertex with ID: %i \n", T1weights[0], T2indexes[0]);
        //---------------

        //ADDING NEW MST EDGE

        //1)Add previously found minimum weight Edge (T1[0] WEIGHT T2[0] EDGE)
        //to the MST by moving this Edge to the first position of R1 R2 R3
        r2[0] = T2indexes[0];  //R2[C] ? 
        r3[0] = T1weights[0];  //R3[C] ?

        //2)Saving current Vertex C= R2[T2[0]]
        c = r2[T2indexes[0]];

        //Copiar datos del host al device que han sido modificados
        hipMemcpy(R2D,r2,(NUMVERTICES-1)*sizeof(int),hipMemcpyDefault);
        hipMemcpy(R3D,r3,(NUMVERTICES-1)*sizeof(int),hipMemcpyDefault);
        hipMemcpy(CD,&c,sizeof(int),hipMemcpyDefault);

        //Kernel 3: Comparing and updating MST
        kernel3<<<bloques, hilos>>>(VGD,VED,R1D,R3D,CD,NED);

        //Copiar datos del device al host
        hipMemcpy(r1,R1D,(NUMVERTICES-1)*sizeof(int),hipMemcpyDefault);
        hipMemcpy(r3,R3D,(NUMVERTICES-1)*sizeof(int),hipMemcpyDefault);

        
        //--------------
        //Recordar que para el print se considera un elemento menos
        //del limite superior ya que realmnete hace el print hasta
        //la posicion indicada
        printf("--- MST ACTUALIZADO ---: \n");
        printf("R1: \n");
        printArrayRange(r1,0,NUMVERTICES-2);
        printf("R2: \n");
        printArrayRange(r2,0,NUMVERTICES-2);
        printf("R3: \n");
        printArrayRange(r3,0,NUMVERTICES-2);
        //--------------

        iCountVer++;

    }//End while
    
    //FIN LOOP |NUMVERTICES|-1 VECES


    //5) Liberar Memoria
    hipFree(VGD);
    hipFree(VED);
    hipFree(R1D);
    hipFree(R2D);
    hipFree(R3D);
    hipFree(T1D);
    hipFree(T2D);
    hipFree(CD);
    hipFree(NED);

    hipEventRecord(stop, 0); 
    hipEventSynchronize(stop);

}//En function primMST
//---- FIN FUNCIONES -----
//Inicio del programa
int main(int argc, char **argv)
{
    setGraph();
    
    //Set root vertex of the MST
    C = 2;

    setVariables();

    printf("IDs threads: \n");
    primMST(VG,EG,R1source,R2destination,R3weigth,C);
    printf("\n");


    printf("Fin del programa V4\n");

}//Fin del main