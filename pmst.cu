
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdlib.h>

//VARIABLES GLOBALES
#define NUMVERTICES 10

//Graph representation with
int *EG;    //Double array of edges |NUMVERTICES|
int *VG;    //Double array of vertices

//------- FUNCIONES --------
void printDoubleArray(int *VX)
{
    int lengthArray = NUMVERTICES;

    for(int i = 0; i <lengthArray; i++)
    {
        printf("%i ", VX[i]);

    }//End for 2

    printf("\n");

    for(int i = lengthArray; i < (lengthArray*2); i++)
    {
         printf("%i ",VX[i]);

    }//End for 2

    printf("\n");

}//Fin funcion printDoubleArray
//------------------------
void printArrayRange(int *VX,int start,int end)
{

    for(int i = start; i <= end; i++)
    {
        printf("%2i ", VX[i]);

    }//End for 2

    printf("\n");

}//Fin funcion printArrayRange
//------------------------
/*
*Function that creates the vaules of the graph
*in the strucutre.
*/
void setGraph()
{
    //Asign and initialize memory for the double array of integers
    //Because is a doble array it is multiplied by 2 only
    VG = (int *) calloc(NUMVERTICES*2, NUMVERTICES*2*sizeof(int) );
    
    int numberEdges = 0;
    int randValue = 0;

    //Inicializacion valores VG
    for (int i = 0; i < NUMVERTICES; i++)
    {
        //Set the index of the Vertex
        VG[i] = i;
        //Set in random way the # of vertices to 
        //wich this vertex is connected

        //#Of vertices can not be 0,becuase all the veritces
        //have to be connected so ensure that at least all
        // the nodes are connected to at leas 2 other vertices
        randValue = rand() % (NUMVERTICES-2) +2;
        VG[i+NUMVERTICES] = randValue;
        //Keep track of the number edges
        numberEdges = numberEdges + randValue;

    }//Fin for 1

    //----------------
    printf("-- Source Vertex --\n");
    printDoubleArray(VG);
    printf("------\n");
    printf("TOTAL EDGES: %i\n",numberEdges);
    //----------------

    //Asign and initialize memory for the double array of integers
    //Because is a doble array it is multiplied by 2 only
    EG = (int *) calloc(numberEdges*2,numberEdges*2*sizeof(int));

    //Initialize EDGE Double array values
    int indxEdges = 0;

    for(int i = 0; i < NUMVERTICES; i++)
    {
        //Num of vertices to wich vertex i has a path
        int numVerticesConn = VG[i+NUMVERTICES];

        //1)Set the destinatio id of the vertex, which can not
        //be repeated and can not be the same as the source
        //vertex i
        int indxDestination = 0;

        //2)Set randomly the value of the weight of edge 1)
        //values of weight from 1 - 100 
        for(int j = 0; j < numVerticesConn; j++)
        {
            //1)
            //Ojo: indxDestination = j a menos que se encuentr
            //que source = destino; en ese caso y por el resto
            //del for j, indxDestination ira uno arriba que j
            if( i == j )
            {
                indxDestination++;
            }//End if

            EG[indxEdges] = indxDestination;

            //2)
            EG[indxEdges+numberEdges] = rand() % (100) +1;

            indxEdges++;
            indxDestination++; 

        }//End for 3

    }//Fin for 2

    //----------------
    printf("-- Destination vertex --\n");
    printArrayRange(EG,0,numberEdges-1);
    printf("-- Weigth of Edge --\n");
    printArrayRange(EG,numberEdges,(numberEdges*2)-1);
    //----------------

}//Fin funcion setGraph
//---- FIN FUNCIONES -----
//Inicio del programa
int main(int argc, char **argv)
{
    setGraph();
    printf("Fin del programa V1\n");

}//Fin del main